#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <functional>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "window.h"
#include "save_image.h"
#include "utils.h"
#include "params.h"

#define ENABLE_CUDA_CHECK 0
#define ENABLE_STREAMS 0


// Kernel Params
struct FractalParams {
    int scr_width;
    int scr_height;
    double fr_x_max;
    double fr_x_min;
    double fr_y_max;
    double fr_y_min;

    FractalParams(const window<int>& scr, const window<double>& fract)
    :   scr_width(scr.width()), scr_height(scr.height()),
        fr_x_max(fract.x_max()), fr_x_min(fract.x_min()),
        fr_y_max(fract.y_max()), fr_y_min(fract.y_min()) {}
};


// Alias for complex type
struct Complex {
    double real;
    double imag;

    __forceinline__ __host__ __device__ Complex(double r=0.0, double i=0.0) : real(r), imag(i) {}

    __forceinline__ __host__ __device__ double magnitude() const {
        return sqrt(real * real + imag * imag);
    }

    __forceinline__ __host__ __device__ Complex operator*(const Complex& other) const {
        return Complex(real * other.real - imag * other.imag, real * other.imag + imag * other.real);
    }

    __forceinline__ __host__ __device__ Complex operator+(const Complex& other) const {
        return Complex(real + other.real, imag + other.imag);    
    }
};

typedef Complex (*fractal_func_t)(Complex, Complex);

__forceinline__ __device__ Complex mandelbrot_func(Complex z, Complex c) {
    return z * z + c;
}

__forceinline__ __device__ Complex triple_mandelbrot_func(Complex z, Complex c) {
    return z * z * z + c;
}

__device__ fractal_func_t fractal_functions[] = {mandelbrot_func, triple_mandelbrot_func};

void checkCudaError(hipError_t result, const char* func, const char* file, int line) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << " code=" << static_cast<unsigned int>(result) 
                  << " \"" << hipGetErrorString(result) << "\" in " << func << std::endl;
        exit(EXIT_FAILURE);
    }
}

#if ENABLE_CUDA_CHECK
    #define CUDA_CHECK(val) checkCudaError((val), #val, __FILE__, __LINE__)
#else
    #define CUDA_CHECK(val) val 
#endif

__host__ void getDeviceProps() {
    int nDevices;
    CUDA_CHECK(hipGetDeviceCount(&nDevices));

    std::cout << "Number of devices: " << nDevices << std::endl;

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        std::cout << "Device Number: " << i << "\n";
        std::cout << "  Device name: " << prop.name << "\n";
        std::cout << "  Memory Clock Rate (MHz): " << prop.memoryClockRate/1024 << "\n";
        std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << "\n";
        std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth/8) / 1.0e6 << "\n";
        std::cout << "  Total global memory (Gbytes): " << static_cast<float>(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0 << "\n";
        std::cout << "  Shared memory per block (Kbytes): " << static_cast<float>(prop.sharedMemPerBlock) / 1024.0 << "\n";
        std::cout << "  minor-major: " << prop.minor << "-" << prop.major << "\n";
        std::cout << "  Warp-size: " << prop.warpSize << "\n";
        std::cout << "  Concurrent kernels: " << (prop.concurrentKernels ? "yes" : "no") << "\n";
        std::cout << "  Concurrent computation/communication: " << (prop.deviceOverlap ? "yes" : "no") << "\n\n";
    }
}

// Convert a pixel coordinate to the complex domain
    __forceinline__ __device__ Complex scale(FractalParams params, Complex c) {
        return Complex(c.real / static_cast<double>(params.scr_width) * (params.fr_x_max - params.fr_x_min) + params.fr_x_min,
                    c.imag / static_cast<double>(params.scr_height) * (params.fr_y_max - params.fr_y_min) + params.fr_y_min);
    }

    // Check if a point is in the set or escapes to infinity, return the number of iterations
    __forceinline__ __device__ int escape(Complex c,fractal_func_t func) {
        Complex z(0);
        int iter = 0;

        while (z.magnitude() < 2.0 && iter < iter_max) {
            z = func(z, c);
            iter++;
        }

        return iter;
    }

    // Loop over each pixel from our image and check if the points associated with this pixel escape to infinity
    __global__ void get_number_iterations(FractalParams params, int *colors, int func_idx) {

        int tix = blockDim.x * blockIdx.x + threadIdx.x;
        int tiy = blockDim.y * blockIdx.y + threadIdx.y;

        if (tiy < params.scr_height && tix < params.scr_width) {
            Complex c(static_cast<double>(tix), static_cast<double>(tiy));
            c = scale(params, c);

            int idx = tiy * params.scr_width + tix;
            fractal_func_t func = fractal_functions[func_idx];
            colors[idx] = escape(c, func);
        }
    }

void fractal(window<int> &scr, window<double> &fract, std::vector<int> &colors,
             int func_idx, const char *fname, bool smooth_color) {
    
    FractalParams params(scr,fract);

    auto start = std::chrono::steady_clock::now();
    int *d_colors;
    CUDA_CHECK(hipMalloc(&d_colors, colors.size() * sizeof(int)));

    dim3 threadsPerBlock(32, 32);	
    dim3 numBlocks((params.scr_width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (params.scr_height + threadsPerBlock.y - 1) / threadsPerBlock.y);

        #if ENABLE_STREAMS
            //TODO
            int N = 8; //number of streams
            hipStream_t streams[N];
            dim3 numBlocksPerStream(numBlocks.x, numBlocks.y / N);

            for (int i = 0; i < N; i++)     
                CUDA_CHECK(hipStreamCreate(&streams[i]));

            for(int i=0;i<N;i++){   
                size_t offx = 0;
                size_t offy = (scr_height/N)*i;
                get_number_iterations<<<numBlocksPerStream,threadsPerBlock,0,streams[i]>>>(params, d_colors+(offy*scr_width+offx),func_idx);
                #if ENABLE_CUDA_CHECK   
                    CUDA_CHECK(hipGetLastError());         
                #endif
                CUDA_CHECK(hipMemcpyAsync(colors.data()+ (offy*scr_width+offx),d_colors+(offy*scr_width+offx), ((scr_height*scr_width/(N*N))*sizeof(int)), hipMemcpyDeviceToHost,streams[i%N]));
            }

            for (int i = 0; i < N; i++) {
                CUDA_CHECK(hipStreamSynchronize(streams[i]));
                CUDA_CHECK(hipStreamDestroy(streams[i]));
            }
        #else

            get_number_iterations<<<numBlocks, threadsPerBlock>>>(params, d_colors, func_idx);
            #if ENABLE_CUDA_CHECK   
                CUDA_CHECK(hipGetLastError());         
            #else
            #endif

            CUDA_CHECK(hipDeviceSynchronize());

            CUDA_CHECK(hipMemcpy(colors.data(), d_colors, colors.size() * sizeof(int), hipMemcpyDeviceToHost));
        #endif
    auto end = std::chrono::steady_clock::now();

    std::cout << "Time to generate " << fname << " = " << std::chrono::duration<double, std::milli>(end - start).count() << " [ms]" << std::endl;
    
    // Save (show) the result as an image
    plot(scr, colors, iter_max, fname, smooth_color);

    // Output details to a text file
    std::ofstream out("./res/fractal_details_GPU.txt", std::ios::app);
    if (out.is_open()) {
        out << "File name: " << fname << "\n";
        out << "GPU Accelerated: true" << "\n";
        out << "Streams: " << (ENABLE_STREAMS ? "enabled" : "disabled") << "\n";
        out << "Debug Mode: " << (ENABLE_CUDA_CHECK ? "enabled" : "disabled") << "\n";
        out << "Time to generate: " << std::chrono::duration<double, std::milli>(end - start).count() << " ms\n";
        out << "Iterations: " << iter_max    << "\n";
        out << "Smooth color: " << (smooth_color ? "true" : "false") << "\n";
        out << "----------------------------------------\n";
        out.close();
    } else {
        std::cerr << "Unable to open file for writing" << std::endl;
    }

    CUDA_CHECK(hipFree(d_colors));
}

void mandelbrot() {
    // Define the size of the image
    window<int> scr(0, win_width, 0, win_height);
    // The domain in which we test for points
    window<double> fract(-2.2, 1.2, -1.7, 1.7);

    const char *fname = "./res/mandelbrot_acc.png";
    bool smooth_color = true;
    std::vector<int> colors(scr.size());

    fractal(scr, fract, colors, 0, fname, smooth_color);
}

void triple_mandelbrot() {
    // Define the size of the image
    window<int> scr(0, win_width, 0, win_height);
    // The domain in which we test for points
    window<double> fract(-1.5, 1.5, -1.5, 1.5);

    const char *fname = "./res/triple_mandelbrot_acc.png";
    bool smooth_color = true;
    std::vector<int> colors(scr.size());

    fractal(scr, fract, colors, 1, fname, smooth_color);
}
void prewarm_gpu(size_t n=1){
    int* dummy;
    hipMalloc(&dummy,n*sizeof(int));
    hipFree(dummy);
    
}
int main() {
    // getDeviceProps();
    prewarm_gpu(win_width * win_height);
    mandelbrot();
    triple_mandelbrot();
}